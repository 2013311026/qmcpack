#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source License.
// See LICENSE file in top directory for details.
//
// Copyright (c) 2018 QMCPACK developers.
//
// File developed by: Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//
// File created by: Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//////////////////////////////////////////////////////////////////////////////////////



#include "determinant_update.h"

template<typename T, int BS>
__global__ void applyW_kernel(const int *delay_list_gpu, const int delay_count,
                                     T* temp_gpu, const int ndelay)
{
  int col = threadIdx.x + blockIdx.x * BS;
  // apply W to temp
  if( col<delay_count ) temp_gpu[ndelay*delay_list_gpu[col] + col] += T(-1);
}

void applyW_cuda(const int *delay_list_gpu, const int delay_count,
                 float* temp_gpu, const int ndelay,
                 hipStream_t& hstream)
{
  const int BS = 128;
  const int NB = (delay_count+BS-1)/BS;
  dim3 dimBlock(BS);
  dim3 dimGrid(NB);
  applyW_kernel<float, BS><<<dimGrid, dimBlock, 0, hstream>>>
  (delay_list_gpu, delay_count, temp_gpu, ndelay);
}

void applyW_cuda(const int *delay_list_gpu, const int delay_count,
                 double* temp_gpu, const int ndelay,
                 hipStream_t& hstream)
{
  const int BS = 128;
  const int NB = (delay_count+BS-1)/BS;
  dim3 dimBlock(BS);
  dim3 dimGrid(NB);
  applyW_kernel<double, BS><<<dimGrid, dimBlock, 0, hstream>>>
  (delay_list_gpu, delay_count, temp_gpu, ndelay);
}


template<typename T, int BS>
__global__ void applyW_stageV_kernel(const int *delay_list_gpu, const int delay_count,
                                     T* temp_gpu, const int numorbs, const int ndelay,
                                     T* V_gpu, const T* Ainv)
{
  int col = threadIdx.x + blockIdx.x * BS;

  // move rows of Ainv to V
  for(int row=0; row<delay_count; row++)
  {
    const T* Ainv_row = Ainv + numorbs * delay_list_gpu[row];
    T* V_row = V_gpu + numorbs * row;
    if( col<numorbs ) V_row[col] = Ainv_row[col];
  }

  // apply W to temp
  if( col<delay_count ) temp_gpu[ndelay*delay_list_gpu[col] + col] += T(-1);
}

void applyW_stageV_cuda(const int *delay_list_gpu, const int delay_count,
                        float* temp_gpu, const int numorbs, const int ndelay,
                        float* V_gpu, const float* Ainv,
                        hipStream_t& hstream)
{
  const int BS = 128;
  const int NB = (numorbs+BS-1)/BS;
  dim3 dimBlock(BS);
  dim3 dimGrid(NB);
  applyW_stageV_kernel<float, BS><<<dimGrid, dimBlock, 0, hstream>>>
  (delay_list_gpu, delay_count, temp_gpu, numorbs, ndelay, V_gpu, Ainv);
}

void applyW_stageV_cuda(const int *delay_list_gpu, const int delay_count,
                        complex<float>* temp_gpu, const int numorbs, const int ndelay,
                        complex<float>* V_gpu, const complex<float>* Ainv,
                        hipStream_t& hstream)
{
  const int BS = 128;
  const int NB = (numorbs+BS-1)/BS;
  dim3 dimBlock(BS);
  dim3 dimGrid(NB);
  applyW_stageV_kernel<hipComplex, BS><<<dimGrid, dimBlock, 0, hstream>>>
  (delay_list_gpu, delay_count, (hipComplex*)temp_gpu, numorbs, ndelay, (hipComplex*)V_gpu, (hipComplex*)Ainv);
}

void applyW_stageV_cuda(const int *delay_list_gpu, const int delay_count,
                        double* temp_gpu, const int numorbs, const int ndelay,
                        double* V_gpu, const double* Ainv,
                        hipStream_t& hstream)
{
  const int BS = 128;
  const int NB = (numorbs+BS-1)/BS;
  dim3 dimBlock(BS);
  dim3 dimGrid(NB);
  applyW_stageV_kernel<double, BS><<<dimGrid, dimBlock, 0, hstream>>>
  (delay_list_gpu, delay_count, temp_gpu, numorbs, ndelay, V_gpu, Ainv);
}

void applyW_stageV_cuda(const int *delay_list_gpu, const int delay_count,
                        complex<double>* temp_gpu, const int numorbs, const int ndelay,
                        complex<double>* V_gpu, const complex<double>* Ainv,
                        hipStream_t& hstream)
{
  const int BS = 128;
  const int NB = (numorbs+BS-1)/BS;
  dim3 dimBlock(BS);
  dim3 dimGrid(NB);
  applyW_stageV_kernel<hipDoubleComplex, BS><<<dimGrid, dimBlock, 0, hstream>>>
  (delay_list_gpu, delay_count, (hipDoubleComplex*)temp_gpu, numorbs, ndelay, (hipDoubleComplex*)V_gpu, (hipDoubleComplex*)Ainv);
}


template<typename T>
__global__ void updateBinv_x_kernel(int* delay_list_gpu,
                                    const int delay_count,
                                    const int rowchanged,
                                    T* Binv_row_gpu,
                                    T* p)
{
  if(threadIdx.x==0)
  {
    delay_list_gpu[delay_count] = rowchanged;
    T y = -p[delay_count];
    for(int i=0; i<delay_count; i++)
      y += Binv_row_gpu[i] * p[i];
    Binv_row_gpu[delay_count] = y = T(1) / y;
    p[delay_count] = -y;
  }
}

void updateBinv_x_cuda(int* delay_list_gpu, const int delay_count,
                       const int rowchanged, float* Binv_row_gpu, float* p,
                       hipStream_t& hstream)
{
  dim3 dimBlock(32);
  dim3 dimGrid(1);
  updateBinv_x_kernel<float><<<dimGrid, dimBlock, 0, hstream>>>
  (delay_list_gpu, delay_count, rowchanged, Binv_row_gpu, p);
}

void updateBinv_x_cuda(int* delay_list_gpu, const int delay_count,
                       const int rowchanged, double* Binv_row_gpu, double* p,
                       hipStream_t& hstream)
{
  dim3 dimBlock(32);
  dim3 dimGrid(1);
  updateBinv_x_kernel<double><<<dimGrid, dimBlock, 0, hstream>>>
  (delay_list_gpu, delay_count, rowchanged, Binv_row_gpu, p);
}

