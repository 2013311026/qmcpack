#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source License.
// See LICENSE file in top directory for details.
//
// Copyright (c) 2018 QMCPACK developers.
//
// File developed by: Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//
// File created by: Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//////////////////////////////////////////////////////////////////////////////////////



#include "determinant_update.h"

template<typename T, int BS>
__global__ void applyW_stageV_kernel(const int *delay_list_gpu, const int delay_count,
                                     T* temp_gpu, const int numorbs, const int ndelay,
                                     T* V_gpu, const T* Ainv)
{
  int col = threadIdx.x + blockIdx.x * BS;

  // move rows of Ainv to V
  for(int row=0; row<delay_count; row++)
  {
    const T* Ainv_row = Ainv + numorbs * delay_list_gpu[row];
    T* V_row = V_gpu + numorbs * row;
    if( col<numorbs ) V_row[col] = Ainv_row[col];
  }

  // apply W to temp
  if( col<delay_count ) temp_gpu[ndelay*delay_list_gpu[col] + col] += T(-1);
}

void applyW_stageV_cuda(const int *delay_list_gpu, const int delay_count,
                        float* temp_gpu, const int numorbs, const int ndelay,
                        float* V_gpu, const float* Ainv,
                        hipStream_t& hstream)
{
  const int BS = 128;
  const int NB = (numorbs+BS-1)/BS;
  dim3 dimBlock(BS);
  dim3 dimGrid(NB);
  applyW_stageV_kernel<float, BS><<<dimGrid, dimBlock, 0, hstream>>>
  (delay_list_gpu, delay_count, temp_gpu, numorbs, ndelay, V_gpu, Ainv);
}

void applyW_stageV_cuda(const int *delay_list_gpu, const int delay_count,
                        double* temp_gpu, const int numorbs, const int ndelay,
                        double* V_gpu, const double* Ainv,
                        hipStream_t& hstream)
{
  const int BS = 128;
  const int NB = (numorbs+BS-1)/BS;
  dim3 dimBlock(BS);
  dim3 dimGrid(NB);
  applyW_stageV_kernel<double, BS><<<dimGrid, dimBlock, 0, hstream>>>
  (delay_list_gpu, delay_count, temp_gpu, numorbs, ndelay, V_gpu, Ainv);
}
