#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source License.
// See LICENSE file in top directory for details.
//
// Copyright (c) 2019 QMCPACK developers.
//
// File developed by: Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//
// File created by: Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//////////////////////////////////////////////////////////////////////////////////////


#include <hip/hip_complex.h>
#include "determinant_update.h"

template<typename T>
__host__ __device__ __inline__ T subtractOne(T x)
{
  return x+T(-1);
}

template<>
__host__ __device__ __inline__ hipComplex subtractOne<hipComplex>(hipComplex x)
{
  return make_hipComplex(hipCrealf(x)-1.0f, hipCimagf(x));
}

template<>
__host__ __device__ __inline__ hipDoubleComplex subtractOne<hipDoubleComplex>(hipDoubleComplex x)
{
  return make_hipDoubleComplex(hipCreal(x)-1.0, hipCimag(x));
}

/** helper kernel for delayed update algorithm
 * W matrix is applied and copy selected rows of Ainv into V
 */
template<typename T, int BS>
__global__ void applyW_stageV_kernel(const int *delay_list_gpu, const int delay_count,
                                     T* temp_gpu, const int numorbs, const int ndelay,
                                     T* V_gpu, const T* Ainv)
{
  int col = threadIdx.x + blockIdx.x * BS;

  // move rows of Ainv to V
  for(int row=0; row<delay_count; row++)
  {
    const T* Ainv_row = Ainv + numorbs * delay_list_gpu[row];
    T* V_row = V_gpu + numorbs * row;
    if( col<numorbs ) V_row[col] = Ainv_row[col];
  }

  // apply W to temp
  if( col<delay_count )
    temp_gpu[ndelay*delay_list_gpu[col] + col] = subtractOne<T>(temp_gpu[ndelay*delay_list_gpu[col] + col]);
}

void applyW_stageV_cuda(const int *delay_list_gpu, const int delay_count,
                        float* temp_gpu, const int numorbs, const int ndelay,
                        float* V_gpu, const float* Ainv,
                        hipStream_t& hstream)
{
  const int BS = 128;
  const int NB = (numorbs+BS-1)/BS;
  dim3 dimBlock(BS);
  dim3 dimGrid(NB);
  applyW_stageV_kernel<float, BS><<<dimGrid, dimBlock, 0, hstream>>>
  (delay_list_gpu, delay_count, temp_gpu, numorbs, ndelay, V_gpu, Ainv);
}

void applyW_stageV_cuda(const int *delay_list_gpu, const int delay_count,
                        std::complex<float>* temp_gpu, const int numorbs, const int ndelay,
                        std::complex<float>* V_gpu, const std::complex<float>* Ainv,
                        hipStream_t& hstream)
{
  const int BS = 128;
  const int NB = (numorbs+BS-1)/BS;
  dim3 dimBlock(BS);
  dim3 dimGrid(NB);
  applyW_stageV_kernel<hipComplex, BS><<<dimGrid, dimBlock, 0, hstream>>>
  (delay_list_gpu, delay_count, (hipComplex*)temp_gpu, numorbs, ndelay, (hipComplex*)V_gpu, (hipComplex*)Ainv);
}

void applyW_stageV_cuda(const int *delay_list_gpu, const int delay_count,
                        double* temp_gpu, const int numorbs, const int ndelay,
                        double* V_gpu, const double* Ainv,
                        hipStream_t& hstream)
{
  const int BS = 128;
  const int NB = (numorbs+BS-1)/BS;
  dim3 dimBlock(BS);
  dim3 dimGrid(NB);
  applyW_stageV_kernel<double, BS><<<dimGrid, dimBlock, 0, hstream>>>
  (delay_list_gpu, delay_count, temp_gpu, numorbs, ndelay, V_gpu, Ainv);
}

void applyW_stageV_cuda(const int *delay_list_gpu, const int delay_count,
                        std::complex<double>* temp_gpu, const int numorbs, const int ndelay,
                        std::complex<double>* V_gpu, const std::complex<double>* Ainv,
                        hipStream_t& hstream)
{
  const int BS = 128;
  const int NB = (numorbs+BS-1)/BS;
  dim3 dimBlock(BS);
  dim3 dimGrid(NB);
  applyW_stageV_kernel<hipDoubleComplex, BS><<<dimGrid, dimBlock, 0, hstream>>>
  (delay_list_gpu, delay_count, (hipDoubleComplex*)temp_gpu, numorbs, ndelay, (hipDoubleComplex*)V_gpu, (hipDoubleComplex*)Ainv);
}

template<typename T>
__host__ __device__ __inline__ T makeZero()
{
  return T(0);
}

template<>
__host__ __device__ __inline__ hipDoubleComplex makeZero<hipDoubleComplex>()
{
  return make_hipDoubleComplex(0.0, 0.0);
}

template<typename T>
__host__ __device__ __inline__ T makeOne()
{
  return T(1);
}

template<>
__host__ __device__ __inline__ hipDoubleComplex makeOne<hipDoubleComplex>()
{
  return make_hipDoubleComplex(1.0, 0.0);
}

template<typename T, int BS>
__global__ void make_identity_matrix_kernel(const int nrows, T* mat, const int lda)
{
  int col = threadIdx.x + blockIdx.x * BS;
  if(col<nrows)
  {
    for(int row = blockIdx.y * BS; row < min((blockIdx.y+1)*BS, nrows); row++)
      mat[row*lda+col] = makeZero<T>();
    if(blockIdx.x==blockIdx.y)
      mat[col*lda+col] = makeOne<T>();
  }
}


void make_identity_matrix_cuda(const int nrows, double* mat, const int lda, hipStream_t& hstream)
{
  const int BS = 128;
  const int NB = (nrows+BS-1)/BS;
  dim3 dimBlock(BS);
  dim3 dimGrid(NB,NB);
  make_identity_matrix_kernel<double, BS><<<dimGrid, dimBlock, 0, hstream>>>
  (nrows, mat, lda);
}

void make_identity_matrix_cuda(const int nrows, std::complex<double>* mat, const int lda, hipStream_t& hstream)
{
  const int BS = 128;
  const int NB = (nrows+BS-1)/BS;
  dim3 dimBlock(BS);
  dim3 dimGrid(NB,NB);
  make_identity_matrix_kernel<hipDoubleComplex, BS><<<dimGrid, dimBlock, 0, hstream>>>
  (nrows, (hipDoubleComplex*)mat, lda);
}
