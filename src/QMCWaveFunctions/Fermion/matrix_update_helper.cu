#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source License.
// See LICENSE file in top directory for details.
//
// Copyright (c) 2020 QMCPACK developers.
//
// File developed by: Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//
// File created by: Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//////////////////////////////////////////////////////////////////////////////////////


#include "QMCWaveFunctions/Fermion/matrix_update_helper.hpp"

namespace qmcplusplus
{
/** interface to cuBLAS_inhouse calls for different data types S/C/D/Z
 */
namespace CUDA
{
template<typename T, int COLBS>
__global__ void copyAinvRow_saveGL_kernel(const int rowchanged,
                                          const int n,
                                          const T* const Ainv[],
                                          const int lda,
                                          T* const temp[],
                                          T* const rcopy[],
                                          const T* const dphi_in[],
                                          const T* const d2phi_in[],
                                          T* const dphi_out[],
                                          T* const d2phi_out[])
{
  const int iw                      = blockIdx.x;
  const T* __restrict__ Ainv_iw     = Ainv[iw];
  T* __restrict__ temp_iw           = temp[iw];
  T* __restrict__ rcopy_iw          = rcopy[iw];
  const T* __restrict__ dphi_in_iw  = dphi_in[iw];
  const T* __restrict__ d2phi_in_iw = d2phi_in[iw];
  T* __restrict__ dphi_out_iw       = dphi_out[iw];
  T* __restrict__ d2phi_out_iw      = d2phi_out[iw];

  const int tid = threadIdx.x;
  if (tid == 0)
    temp_iw[rowchanged] -= T(1);

  const int num_col_blocks = (n + COLBS - 1) / COLBS;
  for (int ib = 0; ib < num_col_blocks; ib++)
  {
    const int col_id = ib * COLBS + threadIdx.x;
    if (col_id < n)
    {
      rcopy_iw[col_id] = Ainv_iw[rowchanged * lda + col_id];

      // the following copying data on the device is not part of SM-1
      // it is intended to copy dphiV and d2phiV from temporary to final without a separate kernel.
      dphi_out_iw[col_id * 3]     = dphi_in_iw[col_id * 3];
      dphi_out_iw[col_id * 3 + 1] = dphi_in_iw[col_id * 3 + 1];
      dphi_out_iw[col_id * 3 + 2] = dphi_in_iw[col_id * 3 + 2];
      d2phi_out_iw[col_id]        = d2phi_in_iw[col_id];
    }
  }
}

hipError_t copyAinvRow_saveGL_cuda(hipStream_t& hstream,
                                    const int rowchanged,
                                    const int n,
                                    const float* const Ainv[],
                                    const int lda,
                                    float* const temp[],
                                    float* const rcopy[],
                                    const float* const dphi_in[],
                                    const float* const d2phi_in[],
                                    float* const dphi_out[],
                                    float* const d2phi_out[],
                                    const int batch_count)
{
  const int COLBS = 32;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count);
  copyAinvRow_saveGL_kernel<float, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(rowchanged, n, Ainv, lda, temp, rcopy,
                                                                             dphi_in, d2phi_in, dphi_out, d2phi_out);
  return hipPeekAtLastError();
}

hipError_t copyAinvRow_saveGL_cuda(hipStream_t& hstream,
                                    const int rowchanged,
                                    const int n,
                                    const double* const Ainv[],
                                    const int lda,
                                    double* const temp[],
                                    double* const rcopy[],
                                    const double* const dphi_in[],
                                    const double* const d2phi_in[],
                                    double* const dphi_out[],
                                    double* const d2phi_out[],
                                    const int batch_count)
{
  const int COLBS = 32;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count);
  copyAinvRow_saveGL_kernel<double, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(rowchanged, n, Ainv, lda, temp, rcopy,
                                                                              dphi_in, d2phi_in, dphi_out, d2phi_out);
  return hipPeekAtLastError();
}

} // namespace CUDA
} // namespace qmcplusplus
