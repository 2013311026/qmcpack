#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source License.
// See LICENSE file in top directory for details.
//
// Copyright (c) 2020 QMCPACK developers.
//
// File developed by: Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//
// File created by: Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//////////////////////////////////////////////////////////////////////////////////////


#include "QMCWaveFunctions/Fermion/matrix_update_helper.hpp"

namespace qmcplusplus
{
/** interface to cuBLAS_inhouse calls for different data types S/C/D/Z
 */
namespace CUDA
{
template<typename T, int COLBS>
__global__ void copyAinvRow_saveGL_kernel(const int rowchanged,
                                          const int n,
                                          const T* const Ainv[],
                                          const int lda,
                                          T* const temp[],
                                          T* const rcopy[],
                                          const T* const dphi_in[],
                                          const T* const d2phi_in[],
                                          T* const dphi_out[],
                                          T* const d2phi_out[])
{
  const int iw                      = blockIdx.x;
  const T* __restrict__ Ainv_iw     = Ainv[iw];
  T* __restrict__ temp_iw           = temp[iw];
  T* __restrict__ rcopy_iw          = rcopy[iw];
  const T* __restrict__ dphi_in_iw  = dphi_in[iw];
  const T* __restrict__ d2phi_in_iw = d2phi_in[iw];
  T* __restrict__ dphi_out_iw       = dphi_out[iw];
  T* __restrict__ d2phi_out_iw      = d2phi_out[iw];

  const int tid = threadIdx.x;
  if (tid == 0)
    temp_iw[rowchanged] -= T(1);

  const int num_col_blocks = (n + COLBS - 1) / COLBS;
  for (int ib = 0; ib < num_col_blocks; ib++)
  {
    const int col_id = ib * COLBS + threadIdx.x;
    if (col_id < n)
    {
      rcopy_iw[col_id] = Ainv_iw[rowchanged * lda + col_id];

      // the following copying data on the device is not part of SM-1
      // it is intended to copy dphiV and d2phiV from temporary to final without a separate kernel.
      dphi_out_iw[col_id * 3]     = dphi_in_iw[col_id * 3];
      dphi_out_iw[col_id * 3 + 1] = dphi_in_iw[col_id * 3 + 1];
      dphi_out_iw[col_id * 3 + 2] = dphi_in_iw[col_id * 3 + 2];
      d2phi_out_iw[col_id]        = d2phi_in_iw[col_id];
    }
  }
}

hipError_t copyAinvRow_saveGL_cuda(hipStream_t& hstream,
                                    const int rowchanged,
                                    const int n,
                                    const float* const Ainv[],
                                    const int lda,
                                    float* const temp[],
                                    float* const rcopy[],
                                    const float* const dphi_in[],
                                    const float* const d2phi_in[],
                                    float* const dphi_out[],
                                    float* const d2phi_out[],
                                    const int batch_count)
{
  if (batch_count == 0)
    return hipSuccess;

  const int COLBS = 64;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count);
  copyAinvRow_saveGL_kernel<float, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(rowchanged, n, Ainv, lda, temp, rcopy,
                                                                             dphi_in, d2phi_in, dphi_out, d2phi_out);
  return hipPeekAtLastError();
}

hipError_t copyAinvRow_saveGL_cuda(hipStream_t& hstream,
                                    const int rowchanged,
                                    const int n,
                                    const double* const Ainv[],
                                    const int lda,
                                    double* const temp[],
                                    double* const rcopy[],
                                    const double* const dphi_in[],
                                    const double* const d2phi_in[],
                                    double* const dphi_out[],
                                    double* const d2phi_out[],
                                    const int batch_count)
{
  if (batch_count == 0)
    return hipSuccess;

  const int COLBS = 64;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count);
  copyAinvRow_saveGL_kernel<double, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(rowchanged, n, Ainv, lda, temp, rcopy,
                                                                              dphi_in, d2phi_in, dphi_out, d2phi_out);
  return hipPeekAtLastError();
}

template<typename T, int COLBS, int DIM = 3>
__global__ void calcGradients_kernel(const int n,
                                     const T* const Ainvrow[],
                                     const T* const dpsiMrow[],
                                     T* const grads_now)
{
  const int iw                    = blockIdx.x;
  const T* __restrict__ invRow    = Ainvrow[iw];
  const T* __restrict__ dpsiM_row = dpsiMrow[iw];

  __shared__ T sum[DIM][COLBS];
  const int tid = threadIdx.x;
  for (int idim = 0; idim < DIM; idim++)
    sum[idim][tid] = T(0);

  const int num_col_blocks = (n + COLBS - 1) / COLBS;
  for (int ib = 0; ib < num_col_blocks; ib++)
  {
    const int col_id = ib * COLBS + tid;
    for (int idim = 0; idim < DIM; idim++)
      if (col_id < n)
        sum[idim][tid] += invRow[col_id] * dpsiM_row[col_id * DIM + idim];
  }

  for (int iend = COLBS / 2; iend > 0; iend /= 2)
  {
    __syncthreads();
    for (int idim = 0; idim < DIM; idim++)
      if (tid < iend)
        sum[idim][tid] += sum[idim][tid + iend];
  }

  if (tid == 0)
    for (int idim = 0; idim < DIM; idim++)
      grads_now[iw * DIM + idim] = sum[idim][0];
}

hipError_t calcGradients_cuda(hipStream_t& hstream,
                               const int n,
                               const float* const Ainvrow[],
                               const float* const dpsiMrow[],
                               float* const grads_now,
                               const int batch_count)
{
  if (batch_count == 0)
    return hipSuccess;

  const int COLBS = 64;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count);
  calcGradients_kernel<float, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(n, Ainvrow, dpsiMrow, grads_now);
  return hipPeekAtLastError();
}

hipError_t calcGradients_cuda(hipStream_t& hstream,
                               const int n,
                               const double* const Ainvrow[],
                               const double* const dpsiMrow[],
                               double* const grads_now,
                               const int batch_count)
{
  if (batch_count == 0)
    return hipSuccess;

  const int COLBS = 64;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count);
  calcGradients_kernel<double, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(n, Ainvrow, dpsiMrow, grads_now);
  return hipPeekAtLastError();
}

template<typename T, int COLBS>
__global__ void add_delay_list_compute_y_kernel(int* const delay_list[],
                                             const int rowchanged,
                                             const int delay_count,
                                             T* const binvrow[],
                                             const T* const p[],
                                             const T* const ratio,
                                             T* const y)
{
  // original CPU code
  //delay_list[delay_count] = rowchanged;
  // x
  //T y = c_ratio;
  //for (int i = 0; i < delay_count; i++)
  //  y += Binv[delay_count][i] * p[i];
  //Binv[delay_count][delay_count] = y = T(1) / y;

  const int tid = threadIdx.x;
  const int iw                    = blockIdx.x;
  int* __restrict__ delay_list_iw    = delay_list[iw];

  if (tid == 0)
    delay_list_iw[delay_count] = rowchanged;

  T* __restrict__ binvrow_iw = binvrow[iw];
  const T* __restrict__ p_iw = p[iw];

  __shared__ T sum[COLBS];
  sum[tid] = T(0);

  const int num_col_blocks = (delay_count + COLBS - 1) / COLBS;
  for (int ib = 0; ib < num_col_blocks; ib++)
  {
    const int col_id = ib * COLBS + tid;
    if (col_id < delay_count)
      sum[tid] += binvrow_iw[col_id] * p_iw[col_id];
  }

  for (int iend = COLBS / 2; iend > 0; iend /= 2)
  {
    __syncthreads();
    if (tid < iend)
      sum[tid] += sum[tid + iend];
  }

  if (tid == 0)
    binvrow_iw[delay_count] = y[iw] = T(1) / (ratio[iw] + sum[0]);
}

hipError_t add_delay_list_compute_y_batched(hipStream_t& hstream,
                                             int* const delay_list[],
                                             const int rowchanged,
                                             const int delay_count,
                                             float* const binvrow[],
                                             const float* const p[],
                                             const float* const ratio,
                                             float* const y,
                                             const int batch_count)
{
  if (batch_count == 0)
    return hipSuccess;

  const int COLBS = 64;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count);
  add_delay_list_compute_y_kernel<float, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(delay_list, rowchanged, delay_count, binvrow, p, ratio, y);
  return hipPeekAtLastError();
}

hipError_t add_delay_list_compute_y_batched(hipStream_t& hstream,
                                             int* const delay_list[],
                                             const int rowchanged,
                                             const int delay_count,
                                             double* const binvrow[],
                                             const double* const p[],
                                             const double* const ratio,
                                             double* const y,
                                             const int batch_count)
{
  if (batch_count == 0)
    return hipSuccess;

  const int COLBS = 64;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count);
  add_delay_list_compute_y_kernel<double, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(delay_list, rowchanged, delay_count, binvrow, p, ratio, y);
  return hipPeekAtLastError();
}

} // namespace CUDA
} // namespace qmcplusplus
