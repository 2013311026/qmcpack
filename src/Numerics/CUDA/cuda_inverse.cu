//////////////////////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source License.
// See LICENSE file in top directory for details.
//
// Copyright (c) 2016 Jeongnim Kim and QMCPACK developers.
//
// File developed by: Ken Esler, kpesler@gmail.com, University of Illinois at Urbana-Champaign
//		      Jeremy McMinnis, jmcminis@gmail.com, University of Illinois at Urbana-Champaign 
//    		      Jeongnim Kim, jeongnim.kim@gmail.com, University of Illinois at Urbana-Champaign
//                    Ying Wai Li, yingwaili@ornl.gov, Oak Ridge National Laboratory
//                    Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//
// File created by:  Ken Esler, kpesler@gmail.com, University of Illinois at Urbana-Champaign
//////////////////////////////////////////////////////////////////////////////////////




// ============ Matrix inversion using cuBLAS library ============ //
//
// To compile as a standalone test program:
//
// 1. Make sure libcublas.so is in the search path
// 2. cd to build/ directory
// 3. For real numbers, compile with
//    nvcc -o cuda_inverse -arch=sm_35 -lcublas -DCUDA_TEST_MAIN
//         ../src/Numerics/CUDA/cuda_inverse.cu
// 
//    For complex numbers, compile with
//    nvcc -o cuda_inverse -arch=sm_35 -lcublas -DCUDA_TEST_MAIN
//         -DQMC_COMPLEX=1 ../src/Numerics/CUDA/cuda_inverse.cu
//
// =============================================================== //

#include <cstdio>
#include <unistd.h>
#include <sstream>
#include <vector>
#include <iostream>
#include <complex>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_complex.h>

#define CONVERT_BS 256
#define INVERSE_BS 16


void
callAndCheckError (hipError_t cudaFunc, const int line)
{
  if (cudaFunc != hipSuccess)
  {
    fprintf(stderr, "CUDA error in %s, line %d \n", __FILE__, line); 
    fprintf(stderr, "CUDA error message : %s \n", hipGetErrorString(cudaFunc));
    fflush(stderr);
    abort();
  }
}

void
callAndCheckError(hipblasStatus_t cublasFunc, const int line)
{
  if (cublasFunc != HIPBLAS_STATUS_SUCCESS)
  {
    fprintf(stderr, "CUBLAS error in %s, line %d \n", __FILE__, line);
    fprintf(stderr, "CUBLAS error message: ");
    switch (cublasFunc)
    {
       case HIPBLAS_STATUS_NOT_INITIALIZED:
         fprintf(stderr, "CUBLAS_STATUS_NOT_INITIALIZED\n");
         break;
       case HIPBLAS_STATUS_ALLOC_FAILED:
         fprintf(stderr, "CUBLAS_STATUS_ALLOC_FAILED\n");
         break;
       case HIPBLAS_STATUS_INVALID_VALUE:
         fprintf(stderr, "CUBLAS_STATUS_INVALID_VALUE\n");
         break;
       case HIPBLAS_STATUS_ARCH_MISMATCH:
         fprintf(stderr, "CUBLAS_STATUS_ARCH_MISMATCH\n");
         break;
       case HIPBLAS_STATUS_MAPPING_ERROR:
         fprintf(stderr, "CUBLAS_STATUS_MAPPING_ERROR\n");
         break;
       case HIPBLAS_STATUS_EXECUTION_FAILED:
         fprintf(stderr, "CUBLAS_STATUS_EXECUTION_FAILED\n");
         break;
       case HIPBLAS_STATUS_INTERNAL_ERROR:
         fprintf(stderr, "CUBLAS_STATUS_INTERNAL_ERROR\n");
         break;
#if (CUDA_VERSION >= 6050)
       case HIPBLAS_STATUS_NOT_SUPPORTED:
         fprintf(stderr, "CUBLAS_STATUS_NOT_SUPPORTED\n");
         break;
       case HIPBLAS_STATUS_UNKNOWN:
         fprintf(stderr, "CUBLAS_STATUS_LICENSE_ERROR\n");
         break;
#endif
       default:
         fprintf(stderr, "unknown\n");
    }
    fflush(stderr);
    abort();
  }
}

// Convert matrix elements from one type (Tsrc) in the source matrix to
// another type (Tdest) and put them in the destination matrix
// (assumed src and dest have the same dimensions)
template <typename Tdest, typename Tsrc>
__global__ void
convert (Tdest **dest_list, Tsrc **src_list, int len)
{
  __shared__ Tsrc *mysrc;
  __shared__ Tdest *mydest;
  if (threadIdx.x == 0)
  {
    mysrc = src_list[blockIdx.y];
    mydest = dest_list[blockIdx.y];
  }
  __syncthreads();
  int i = blockIdx.x * CONVERT_BS + threadIdx.x;
  if (i < len)
    mydest[i] = (Tdest) mysrc[i];
}

// Convert for complex numbers
template <typename Tdest, typename Tdest2, typename Tsrc>
__global__ void
convert_complex (Tdest **dest_list, Tsrc **src_list, int len)
{
  __shared__ Tsrc *mysrc;
  __shared__ Tdest *mydest;
  if (threadIdx.x == 0)
  {
    mysrc = src_list[blockIdx.y];
    mydest = dest_list[blockIdx.y];
  }
  __syncthreads();
  int i = blockIdx.x * CONVERT_BS + threadIdx.x;
  if (i < len) {
    mydest[i].x = (Tdest2) mysrc[i].x;
    mydest[i].y = (Tdest2) mysrc[i].y;
  }
}

// C = A - B
template <typename T>
__global__ void
subtract (T **C, T **A, T **B, int len)
{
  __shared__ T *myA, *myB, *myC;
  if (threadIdx.x == 0)
  {
    myA = A[blockIdx.y];
    myB = B[blockIdx.y];
    myC = C[blockIdx.y];
  }
  __syncthreads();
  int i = blockIdx.x * CONVERT_BS + threadIdx.x;
  if (i < len)
    myC[i] = myA[i] - myB[i];
}

/** Calculate Lemma Matrix: I_k + V' * ( A^(-1) * U )
  * for each walker
  * -> returns L-U decomposed lemma matrix for easy determinant calculations and inverse calculation later
  */
void
cublas_lemma_mats (hipblasHandle_t handle,
                   float *AList_d[], float *AWorkList_d[],
                   float *AinvList_d[], float *AinvkList_d[], float *U_d[],
                   float *lemma_d[], float *AinvUList_d[],
                   int k, int N, int nw, int RowStride)
{
  float one=1.0;
  float zero=0.0;
  // Calculate Lemma Matrix
  // V^-1 * A^(-1) * U
  // per walker: [k x N] * [N x k] = [k x k]
  callAndCheckError( hipblasSgemmBatched( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, k, k, N,
                                         &one,
                                         (const float**)AinvkList_d, RowStride,
                                         (const float**)U_d, RowStride, &zero,
                                         lemma_d, k,
                                         nw), __LINE__ );
  // Calculate - A^-1*dU
  dim3 dimBlockConvert (CONVERT_BS);
  dim3 dimGridConvert ((k*RowStride + (CONVERT_BS-1)) / CONVERT_BS, nw);
  // Calculate -dU=U(old)-U(new)
  subtract <<< dimGridConvert, dimBlockConvert >>> (AWorkList_d, AList_d, U_d, k*RowStride);
  // -A^(-1) * dU
  // per walker: [N x N] * [N x k] = [N x k]
#ifndef AINVU_TRANSPOSE
  callAndCheckError( hipblasSgemmBatched( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, k, N,
                                         &one,
                                         (const float**)AinvList_d, RowStride,
                                         (const float**)AWorkList_d, RowStride, &zero,
                                         AinvUList_d, RowStride,
                                         nw), __LINE__ );
#else
  // calculate AinvU as row major
  // per walker: [N x k]^T * [N x N]^T = [k x N] * [N x N] = [k x N]
  callAndCheckError( cublasSgemmBatched( handle, CUBLAS_OP_T, CUBLAS_OP_T, k, N, N,
                                         &one,
                                         (const float**)AWorkList_d, RowStride,
                                         (const float**)AinvList_d, RowStride, &zero,
                                         AinvUList_d, k,
                                         nw), __LINE__ );
#endif
//  cudaDeviceSynchronize();
}

void
cublas_ainv_row (hipblasHandle_t handle,
                 float *AinvkList_d[], float *AWorkList_d[], float *AinvList_d[],
                 int k, int N, int nw, int RowStride)
{
  float one=1.0;
  // A^-1 - { A^-1 * dU  * Lemma^-1 } * { V' * A^(-1) }
  // per walker: [1 x N] - [1 x k] * [k x N] = [1 x N]
  callAndCheckError( hipblasSgemmBatched( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, N, k,
                                         &one,
                                         (const float**)AWorkList_d, 1,
                                         (const float**)AinvkList_d, RowStride, &one,
                                         AinvList_d, 1,
                                         nw), __LINE__ );
}

void
cublas_ainv_row (hipblasHandle_t handle,
                 double *AinvkList_d[], double *AWorkList_d[], double *AinvList_d[],
                 int k, int N, int nw, int RowStride)
{
  double one=1.0;
  // A^-1 - { A^-1 * dU  * Lemma^-1 } * { V' * A^(-1) }
  // per walker: [1 x N] - [1 x k] * [k x N] = [1 x N]
  callAndCheckError( hipblasDgemmBatched( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, N, k,
                                         &one,
                                         (const double**)AWorkList_d, 1,
                                         (const double**)AinvkList_d, RowStride, &one,
                                         AinvList_d, 1,
                                         nw), __LINE__ );
}


// #define DEBUG_DELAYED
// #define USE_TRSM

void
cublas_smw_update (hipblasHandle_t handle,
                   float *AinvkList_d[], float *AinvList_d[],
                   float *AinvUList_d[], float *AWorkList_d[],
                   float *lemma_inv[], float *lemma_lu[],
                   int *infoArray,
                   int k, int kd, int M, int N, int nw, int RowStride)
{
#ifdef DEBUG_DELAYED
  fprintf(stderr,"*** Sherman-Morrison-Woodbury Update (k = %i, %i walkers) ***\n",k,nw);
#endif
  int pitch=RowStride;
  if(M==1) pitch=1;
  float one=1.0;

  // LU decomposition needs to be updated
  callAndCheckError( hipblasSgetrfBatched( handle, k, lemma_lu, kd, NULL,
                                          infoArray, nw), __LINE__ );

#ifdef USE_TRSM
  if(M==1)
  {
    // {-A^-1 * dU } * Lemma^(-1) => solve for y: Lemma y * (L * U) = (y * L) * U = -A^-1 * dU
    // z * U = -A^-1 *dU
    callAndCheckError( cublasStrsmBatched( handle, CUBLAS_SIDE_RIGHT, CUBLAS_FILL_MODE_UPPER, CUBLAS_OP_N, CUBLAS_DIAG_NON_UNIT,
                                           M, k, &one, (const float**) lemma_lu, kd,
                                           AWorkList_d, pitch, nw), __LINE__ );
    // y * L = z => y = {-A^-1 * dU } * Lemma^(-1)
    callAndCheckError( cublasStrsmBatched( handle, CUBLAS_SIDE_RIGHT, CUBLAS_FILL_MODE_LOWER, CUBLAS_OP_N, CUBLAS_DIAG_UNIT,
                                           M, k, &one, (const float**) lemma_lu, kd,
                                           AWorkList_d, pitch, nw), __LINE__ );
    // A^-1 + { -A^-1 * dU *  Lemma^-1 } * { V' * A^(-1) }
    // per walker: [1 x N] - [1 x k] * [k x N] = [1 x N]
    callAndCheckError( cublasSgemmBatched( handle, CUBLAS_OP_N, CUBLAS_OP_N, M, N, k,
                                           &one,
                                           (const float**)AWorkList_d, M,
                                           (const float**)AinvkList_d, RowStride, &one,
                                           AinvList_d, pitch,
                                           nw), __LINE__ );
  } else
  {
    // Lemma^(-1) * V' * A^(-1) => solve for y: Lemma (L * U) * y = L * (U * y) = V' * A^(-1)
    // L * z = V' * A^(-1)
    callAndCheckError( cublasStrsmBatched( handle, CUBLAS_SIDE_LEFT, CUBLAS_FILL_MODE_LOWER, CUBLAS_OP_N, CUBLAS_DIAG_UNIT,
                                           k, N, &one, (const float**) lemma_lu, kd,
                                           AWorkList_d, k, nw), __LINE__ );
    // U * y = z => y = Lemma^(-1) * V' * A^(-1)
    callAndCheckError( cublasStrsmBatched( handle, CUBLAS_SIDE_LEFT, CUBLAS_FILL_MODE_UPPER, CUBLAS_OP_N, CUBLAS_DIAG_NON_UNIT,
                                           k, N, &one, (const float**) lemma_lu, kd,
                                           AWorkList_d, k, nw), __LINE__ );
    // A^-1 + { -A^-1 * dU } * { Lemma^-1 * V' * A^(-1) }
    // per walker: [M x N] - [M x k] * [k x N] = [M x N]
    callAndCheckError( cublasSgemmBatched( handle, CUBLAS_OP_N, CUBLAS_OP_N, M, N, k,
                                           &one,
                                           (const float**)AinvUList_d, RowStride,
                                           (const float**)AWorkList_d, k, &one,
                                           AinvList_d, pitch,
                                           nw), __LINE__ );
  }
#else
  float zero=0.0;
  // Calculate Lemma Inverse and store it in lemma_d
  // per walker: [k x k]^-1 = [k x k]
  callAndCheckError( hipblasSgetriBatched( handle, k, (const float**) lemma_lu, kd, NULL,
                                          lemma_inv, k, infoArray, nw), __LINE__ );
  // Calculate new A inverse using Sherman-Morrison-Woodbury formula
  if(M==1) // row update can use different order to save flops
  {
    // { -A^-1 * dU } * Lemma^-1
    // per walker: [M x k] * [k x k] = [M x k]
    callAndCheckError( hipblasSgemmBatched( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, k, k,
                                           &one,
                                           (const float**)AinvUList_d, RowStride,
                                           (const float**)lemma_inv, k, &zero,
                                           AWorkList_d, M,
                                           nw), __LINE__ );
    // A^-1 - { A^-1 * dU  * Lemma^-1 } * { V' * A^(-1) }
    // per walker: [M x N] - [M x k] * [k x N] = [M x N]
    callAndCheckError( hipblasSgemmBatched( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, k,
                                           &one,
                                           (const float**)AWorkList_d, M,
                                           (const float**)AinvkList_d, RowStride, &one,
                                           AinvList_d, pitch,
                                           nw), __LINE__ );
  } else
  {
    // Need to use this matrix order for the overall update as AinvList and AinvkList have overlapping memory
    // Lemma^-1 * V' * A^(-1)
    // per walker: [k x k] * [k x N] = [k x N]
    callAndCheckError( hipblasSgemmBatched( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, k, N, k,
                                           &one,
                                           (const float**)lemma_inv, k,
                                           (const float**)AinvkList_d, RowStride, &zero,
                                           AWorkList_d, k,
                                           nw), __LINE__ );
    // A^-1 + { -A^-1 * dU } * { Lemma^-1 * V' * A^(-1) }
    // per walker: [M x N] - [M x k] * [k x N] = [M x N]
    callAndCheckError( hipblasSgemmBatched( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, k,
                                           &one,
                                           (const float**)AinvUList_d, RowStride,
                                           (const float**)AWorkList_d, k, &one,
                                           AinvList_d, pitch,
                                           nw), __LINE__ );
  }
#endif
}

/** Calculate Lemma Matrix: I_k + V' * ( A^(-1) * U )
  * for each walker
  * -> returns L-U decomposed lemma matrix for easy determinant calculations and inverse calculation later
  */
void
cublas_lemma_mats (hipblasHandle_t handle,
                   double *AList_d[], double *AWorkList_d[],
                   double *AinvList_d[], double *AinvkList_d[], double *U_d[],
                   double *lemma_d[], double *AinvUList_d[],
                   int k, int N, int nw, int RowStride)
{
  double one=1.0;
  double zero=0.0;
  // Calculate Lemma Matrix
  // V^-1 * A^(-1) * U
  // per walker: [k x N] * [N x k] = [k x k]
  callAndCheckError( hipblasDgemmBatched( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, k, k, N,
                                         &one,
                                         (const double**)AinvkList_d, RowStride,
                                         (const double**)U_d, RowStride, &zero,
                                         lemma_d, k,
                                         nw), __LINE__ );
  // Calculate - A^-1*dU
  dim3 dimBlockConvert (CONVERT_BS);
  dim3 dimGridConvert ((k*RowStride + (CONVERT_BS-1)) / CONVERT_BS, nw);
  // Calculate -dU=U(old)-U(new)
  subtract <<< dimGridConvert, dimBlockConvert >>> (AWorkList_d, AList_d, U_d, k*RowStride);
  // -A^(-1) * dU
  // per walker: [N x N] * [N x k] = [N x k]
  callAndCheckError( hipblasDgemmBatched( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, k, N,
                                         &one,
                                         (const double**)AinvList_d, RowStride,
                                         (const double**)AWorkList_d, RowStride, &zero,
                                         AinvUList_d, RowStride,
                                         nw), __LINE__ );
  // no synchronization needed here as this function call will always be followed by lemma lu calculation
}

void
cublas_smw_update (hipblasHandle_t handle,
                   double *AinvkList_d[], double *AinvList_d[], 
                   double *AinvUList_d[], double *AWorkList_d[],
                   double *lemma_inv[], double *lemma_lu[],
                   int *infoArray,
                   int k, int kd, int M, int N, int nw, int RowStride)
{
#ifdef DEBUG_DELAYED
  fprintf(stderr,"*** Sherman-Morrison-Woodbury Update (k = %i, %i walkers) ***\n",k,nw);
#endif
  int pitch=RowStride;
  if(M==1) pitch=1;
  double one=1.0;

  // LU decomposition needs to be updated
  callAndCheckError( hipblasDgetrfBatched( handle, k, lemma_lu, kd, NULL,
                                          infoArray, nw), __LINE__ );

#ifdef USE_TRSM
  if(M==1)
  {
    // {-A^-1 * dU } * Lemma^(-1) => solve for y: Lemma y * (L * U) = (y * L) * U = -A^-1 * dU
    // z * U = -A^-1 *dU
    callAndCheckError( cublasDtrsmBatched( handle, CUBLAS_SIDE_RIGHT, CUBLAS_FILL_MODE_UPPER, CUBLAS_OP_N, CUBLAS_DIAG_NON_UNIT,
                                           M, k, &one, (const double**) lemma_lu, kd,
                                           AWorkList_d, pitch, nw), __LINE__ );
    // y * L = z => y = {-A^-1 * dU } * Lemma^(-1)
    callAndCheckError( cublasDtrsmBatched( handle, CUBLAS_SIDE_RIGHT, CUBLAS_FILL_MODE_LOWER, CUBLAS_OP_N, CUBLAS_DIAG_UNIT,
                                           M, k, &one, (const double**) lemma_lu, kd,
                                           AWorkList_d, pitch, nw), __LINE__ );
    // A^-1 + { -A^-1 * dU *  Lemma^-1 } * { V' * A^(-1) }
    // per walker: [1 x N] - [1 x k] * [k x N] = [1 x N]
    callAndCheckError( cublasDgemmBatched( handle, CUBLAS_OP_N, CUBLAS_OP_N, M, N, k,
                                           &one,
                                           (const double**)AWorkList_d, M,
                                           (const double**)AinvkList_d, RowStride, &one,
                                           AinvList_d, pitch,
                                           nw), __LINE__ );
  } else
  {
    // Lemma^(-1) * V' * A^(-1) => solve for y: Lemma (L * U) * y = L * (U * y) = V' * A^(-1)
    // L * z = V' * A^(-1)
    callAndCheckError( cublasDtrsmBatched( handle, CUBLAS_SIDE_LEFT, CUBLAS_FILL_MODE_LOWER, CUBLAS_OP_N, CUBLAS_DIAG_UNIT,
                                           k, N, &one, (const double**) lemma_lu, kd,
                                           AWorkList_d, k, nw), __LINE__ );
    // U * y = z => y = Lemma^(-1) * V' * A^(-1)
    callAndCheckError( cublasDtrsmBatched( handle, CUBLAS_SIDE_LEFT, CUBLAS_FILL_MODE_UPPER, CUBLAS_OP_N, CUBLAS_DIAG_NON_UNIT,
                                           k, N, &one, (const double**) lemma_lu, kd,
                                           AWorkList_d, k, nw), __LINE__ );
    // A^-1 + { -A^-1 * dU } * { Lemma^-1 * V' * A^(-1) }
    // per walker: [M x N] - [M x k] * [k x N] = [M x N]
    callAndCheckError( cublasDgemmBatched( handle, CUBLAS_OP_N, CUBLAS_OP_N, M, N, k,
                                           &one,
                                           (const double**)AinvUList_d, RowStride,
                                           (const double**)AWorkList_d, k, &one,
                                           AinvList_d, pitch,
                                           nw), __LINE__ );
  }
#else
  double zero=0.0;
  // Calculate Lemma Inverse and store it in lemma_d
  // per walker: [k x k]^-1 = [k x k]
  callAndCheckError( hipblasDgetriBatched( handle, k, (const double**) lemma_lu, kd, NULL,
                                          lemma_inv, k, infoArray, nw), __LINE__ );
  // Calculate new A inverse using Sherman-Morrison-Woodbury formula
  if(M==1) // row update can use different order to save flops
  {
    // { -A^-1 * dU } * Lemma^-1
    // per walker: [M x k] * [k x k] = [M x k]
    callAndCheckError( hipblasDgemmBatched( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, k, k,
                                           &one,
                                           (const double**)AinvUList_d, RowStride,
                                           (const double**)lemma_inv, k, &zero,
                                           AWorkList_d, M,
                                           nw), __LINE__ );
    // A^-1 - { A^-1 * dU  * Lemma^-1 } * { V' * A^(-1) }
    // per walker: [M x N] - [M x k] * [k x N] = [M x N]
    callAndCheckError( hipblasDgemmBatched( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, k,
                                           &one,
                                           (const double**)AWorkList_d, M,
                                           (const double**)AinvkList_d, RowStride, &one,
                                           AinvList_d, pitch,
                                           nw), __LINE__ );
  } else
  {
    // Need to use this matrix order for the overall update as AinvList and AinvkList have overlapping memory
    // Lemma^-1 * V' * A^(-1)
    // per walker: [k x k] * [k x N] = [k x N]
    callAndCheckError( hipblasDgemmBatched( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, k, N, k,
                                           &one,
                                           (const double**)lemma_inv, k,
                                           (const double**)AinvkList_d, RowStride, &zero,
                                           AWorkList_d, k,
                                           nw), __LINE__ );
    // A^-1 + { -A^-1 * dU } * { Lemma^-1 * V' * A^(-1) }
    // per walker: [M x N] - [M x k] * [k x N] = [M x N]
    callAndCheckError( hipblasDgemmBatched( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, k,
                                           &one,
                                           (const double**)AinvUList_d, RowStride,
                                           (const double**)AWorkList_d, k, &one,
                                           AinvList_d, pitch,
                                           nw), __LINE__ );
  }
#endif
}

// Four matrix inversion functions
// 1. for float matrices
//    useHigherPrecision = false --> single precision operations
//    useHigherPrecision = true  --> double precision operations  (default)
void
cublas_inverse (hipblasHandle_t handle, 
                float *Alist_d[], float *Ainvlist_d[],
                float *AWorklist_d[], float *AinvWorklist_d[],
                int *PivotArray, int *infoArray,
                int N, int rowStride, int numMats,
                bool useHigherPrecision)
{

  // Info array tells if a matrix inversion is successful
  // = 0 : successful
  // = k : U(k,k) = 0; inversion failed 

  // If double precision operations are desired...
  if (useHigherPrecision)
  {

    // (i)   convert elements in Alist from float to double, put them in AWorklist
    dim3 dimBlockConvert (CONVERT_BS);
    dim3 dimGridConvert ((N*rowStride + (CONVERT_BS-1)) / CONVERT_BS, numMats);
    convert <<< dimGridConvert, dimBlockConvert >>> ((double**)AWorklist_d, Alist_d, N*rowStride);

    // (ii)  call cublas to do matrix inversion
    //       LU decomposition
    callAndCheckError( hipblasDgetrfBatched( handle, N, (double**)AWorklist_d, rowStride, PivotArray,
                                            infoArray, numMats), __LINE__ );

    //       Inversion
#if (CUDA_VERSION >= 6050)
    callAndCheckError( hipblasDgetriBatched( handle, N, (const double**)AWorklist_d, rowStride, PivotArray,
                                            (double**)AinvWorklist_d, rowStride, infoArray+numMats, numMats), __LINE__ );
#else
    callAndCheckError( cublasDgetriBatched( handle, N, (double**)AWorklist_d, rowStride, PivotArray,
                                            (double**)AinvWorklist_d, rowStride, infoArray+numMats, numMats), __LINE__ );
#endif

    // (iii) convert results back to single precision
    convert <<< dimGridConvert, dimBlockConvert >>> (Ainvlist_d, (double**)AinvWorklist_d, N*rowStride);

  }
  // else, carry out single precision operations
  else
  {
    // Call cublas to do matrix inversion
    // LU decomposition
    callAndCheckError( hipblasSgetrfBatched( handle, N, Alist_d, rowStride, PivotArray,
                                            infoArray, numMats), __LINE__ );
  
    // Inversion
#if (CUDA_VERSION >= 6050)
    callAndCheckError( hipblasSgetriBatched( handle, N, (const float**) Alist_d, rowStride, PivotArray,
                                            Ainvlist_d, rowStride, infoArray+numMats, numMats), __LINE__ );
#else
    callAndCheckError( cublasSgetriBatched( handle, N, Alist_d, rowStride, PivotArray,
                                            Ainvlist_d, rowStride, infoArray+numMats, numMats), __LINE__ );
#endif
  }

  hipDeviceSynchronize();
}

// 2. for double matrices
void
cublas_inverse (hipblasHandle_t handle, 
                double *Alist_d[], double *Ainvlist_d[],
                double *AWorklist_d[], double *AinvWorklist_d[],
                int *PivotArray, int *infoArray,
                int N, int rowStride, int numMats,
                bool useHigherPrecision)
{

  // Info array tells if a matrix inversion is successful
  // = 0 : successful
  // = k : U(k,k) = 0; inversion failed 

  // (i)   copy all the elements of Alist to AWorklist
  dim3 dimBlockConvert (CONVERT_BS);
  dim3 dimGridConvert ((N*rowStride + (CONVERT_BS-1)) / CONVERT_BS, numMats);
  convert <<< dimGridConvert, dimBlockConvert >>> (AWorklist_d, Alist_d, N*rowStride);
  
  // (ii)  call cublas functions to do inversion
  //       LU decomposition
  callAndCheckError( hipblasDgetrfBatched( handle, N, AWorklist_d, rowStride, PivotArray,
                                          infoArray, numMats), __LINE__ );

  //       Inversion
#if (CUDA_VERSION >= 6050)
  callAndCheckError( hipblasDgetriBatched( handle, N, (const double**) AWorklist_d, rowStride, PivotArray,
                                          Ainvlist_d, rowStride, infoArray+numMats, numMats), __LINE__ );
#else
  callAndCheckError( cublasDgetriBatched( handle, N, AWorklist_d, rowStride, PivotArray,
                                          Ainvlist_d, rowStride, infoArray+numMats, numMats), __LINE__ );
#endif

  hipDeviceSynchronize();
}

// 3. for complex float matrices
//    useHigherPrecision = false --> single precision operations
//    useHigherPrecision = true  --> double precision operations  (default)
void
cublas_inverse (hipblasHandle_t handle, 
                std::complex<float> *Alist_d[], std::complex<float> *Ainvlist_d[],
                std::complex<float> *AWorklist_d[], std::complex<float> *AinvWorklist_d[],
                int *PivotArray, int *infoArray,
                int N, int rowStride, int numMats,
                bool useHigherPrecision)
{

  // Info array tells if a matrix inversion is successful
  // = 0 : successful
  // = k : U(k,k) = 0; inversion failed 

  // If double precision operations are desired...
  if (useHigherPrecision)
  {

    // (i)   convert elements in Alist from float to double, put them in AWorklist
    dim3 dimBlockConvert (CONVERT_BS);
    dim3 dimGridConvert ((N*rowStride + (CONVERT_BS-1)) / CONVERT_BS, numMats);
    convert_complex<hipDoubleComplex, double, hipComplex> <<< dimGridConvert, dimBlockConvert >>> ((hipDoubleComplex**)AWorklist_d, (hipComplex**)Alist_d, N*rowStride);

    // (ii)  call cublas to do matrix inversion
    //       LU decomposition
    callAndCheckError( hipblasZgetrfBatched( handle, N, (hipDoubleComplex**)AWorklist_d, rowStride, PivotArray, infoArray, numMats), __LINE__ );
    //       Inversion
#if (CUDA_VERSION >= 6050)
    callAndCheckError( hipblasZgetriBatched( handle, N, (const hipDoubleComplex**)AWorklist_d, rowStride, PivotArray, (hipDoubleComplex**)AinvWorklist_d, rowStride, infoArray+numMats, numMats), __LINE__ );
#else
    callAndCheckError( cublasZgetriBatched( handle, N, (cuDoubleComplex**)AWorklist_d, rowStride, PivotArray, (cuDoubleComplex**)AinvWorklist_d, rowStride, infoArray+numMats, numMats), __LINE__ );
#endif

    // (iii) convert results back to single precision
    convert_complex<hipComplex, float, hipDoubleComplex> <<< dimGridConvert, dimBlockConvert >>> ((hipComplex**)Ainvlist_d, (hipDoubleComplex**)AinvWorklist_d, N*rowStride);

  }
  // else, carry out single precision operations
  else
  {
    // Call cublas to do matrix inversion
    // LU decomposition
    callAndCheckError( hipblasCgetrfBatched( handle, N, (hipComplex**)Alist_d, rowStride, PivotArray,
                                            infoArray, numMats), __LINE__ );
  
    // Inversion
#if (CUDA_VERSION >= 6050)
    callAndCheckError( hipblasCgetriBatched( handle, N, (const hipComplex**)Alist_d, rowStride, PivotArray, (hipComplex**)Ainvlist_d, rowStride, infoArray+numMats, numMats), __LINE__ );
#else
    callAndCheckError( cublasCgetriBatched( handle, N, (cuComplex**)Alist_d, rowStride, PivotArray, (cuComplex**)Ainvlist_d, rowStride, infoArray+numMats, numMats), __LINE__ );
#endif
  }

  hipDeviceSynchronize();
}

// 4. for complex double matrices
void
cublas_inverse (hipblasHandle_t handle, 
                std::complex<double> *Alist_d[], std::complex<double> *Ainvlist_d[],
                std::complex<double> *AWorklist_d[], std::complex<double> *AinvWorklist_d[],
                int *PivotArray, int *infoArray,
                int N, int rowStride, int numMats,
                bool useHigherPrecision)
{

  // Info array tells if a matrix inversion is successful
  // = 0 : successful
  // = k : U(k,k) = 0; inversion failed 

  // (i)   copy all the elements of Alist to AWorklist
  dim3 dimBlockConvert (CONVERT_BS);
  dim3 dimGridConvert ((N*rowStride + (CONVERT_BS-1)) / CONVERT_BS, numMats);
  convert_complex<hipDoubleComplex, double, hipDoubleComplex> <<< dimGridConvert, dimBlockConvert >>> ((hipDoubleComplex**)AWorklist_d, (hipDoubleComplex**)Alist_d, N*rowStride);

  // (ii)  call cublas to do matrix inversion
  //       LU decomposition
  callAndCheckError( hipblasZgetrfBatched( handle, N, (hipDoubleComplex**)AWorklist_d, rowStride, PivotArray, infoArray, numMats), __LINE__ );
  //       Inversion
#if (CUDA_VERSION >= 6050)
  callAndCheckError( hipblasZgetriBatched( handle, N, (const hipDoubleComplex**)AWorklist_d, rowStride, PivotArray, (hipDoubleComplex**)Ainvlist_d, rowStride, infoArray+numMats, numMats), __LINE__ );
#else
  callAndCheckError( cublasZgetriBatched( handle, N, (cuDoubleComplex**)AWorklist_d, rowStride, PivotArray, (cuDoubleComplex**)Ainvlist_d, rowStride, infoArray+numMats, numMats), __LINE__ );
#endif

  hipDeviceSynchronize();
}



//////////////////////////////////////////////////////
//                  Test routines                   //
//////////////////////////////////////////////////////

#ifdef CUDA_TEST_MAIN

template<typename T>
void
test_cublas_inverse(int matSize, int numMats)
{

  // Initialize cublas
  cublasHandle_t handle;
  callAndCheckError( cublasCreate(&handle), __LINE__);

  srand48((long) 12394);
  int N = matSize;
  int row_stride = (matSize+15) / 16 * 16;
  T **Alist, **AWorklist;
  T **Alist_d, **AWorklist_d;
  T **Clist, **CWorklist;
  T **Clist_d, **CWorklist_d;

  // Allocate arrays of pointers (one set on host, one set on device)
  // pointing to the starting address (on device) of each matrix and its buffer
  // (similar to DiracDeterminantCUDA)
  Alist = (T**) malloc(numMats * sizeof(T*));
  callAndCheckError( cudaMalloc((void**) &Alist_d, numMats * sizeof(T*)), __LINE__ );

  AWorklist = (T**) malloc(numMats * sizeof(T*));
  callAndCheckError( cudaMalloc((void**) &AWorklist_d, numMats * sizeof(T*)), __LINE__ );

  Clist = (T**) malloc(numMats * sizeof(T*));
  callAndCheckError( cudaMalloc((void**) &Clist_d, numMats * sizeof(T*)), __LINE__ );

  CWorklist = (T**) malloc(numMats * sizeof(T*));
  callAndCheckError( cudaMalloc((void**) &CWorklist_d, numMats * sizeof(T*)), __LINE__ );

  // Generate matrices filled with random numbers
  T* A = (T*) malloc(sizeof(T) * numMats * N * row_stride);

  for (int j=0; j<numMats; j++)
    for (int i=0; i<N*row_stride; i++) {
#ifndef QMC_COMPLEX
        A[j*N*row_stride+i] = 1.0 * (drand48() - 0.5);
#else
        A[j*N*row_stride+i] = T(1.0 * (drand48() - 0.5), 1.0 * (drand48() - 0.5));
#endif
    }

  // Allocate memory on device for each matrix
  for (int mat=0; mat<numMats; mat++)
  {
    callAndCheckError( cudaMalloc((void**) &(Alist[mat]), N * row_stride * sizeof(T)), __LINE__ );

    callAndCheckError( cudaMemcpyAsync(Alist[mat], &A[mat*N*row_stride],
                                       N * row_stride * sizeof(T),
                                       cudaMemcpyHostToDevice), __LINE__ );

    callAndCheckError( cudaMalloc((void**) &(AWorklist[mat]), 2 * N * row_stride * sizeof(T)), __LINE__ );

    callAndCheckError( cudaMalloc((void**) &(Clist[mat]), N * row_stride * sizeof(T)), __LINE__ );

    callAndCheckError( cudaMalloc((void**) &(CWorklist[mat]), 2 * N * row_stride * sizeof(T)), __LINE__ );
  }

  // Copy the starting address of each matrix
  callAndCheckError( cudaMemcpyAsync (Alist_d, Alist, numMats * sizeof(T*),
                                      cudaMemcpyHostToDevice), __LINE__ ); 

  callAndCheckError( cudaMemcpyAsync (AWorklist_d, AWorklist, numMats * sizeof(T*),
                                      cudaMemcpyHostToDevice), __LINE__ );

  callAndCheckError( cudaMemcpyAsync (Clist_d, Clist, numMats * sizeof(T*),
                                      cudaMemcpyHostToDevice), __LINE__ );

  callAndCheckError( cudaMemcpyAsync (CWorklist_d, CWorklist, numMats * sizeof(T*),
                                      cudaMemcpyHostToDevice), __LINE__ );

  cudaDeviceSynchronize();
 
  clock_t start = clock();
 
  // Call cublas functions to do inversion
  cublas_inverse (handle, Alist_d, Clist_d, AWorklist_d, CWorklist_d, N, row_stride, numMats, true);
  cudaDeviceSynchronize();
  
  clock_t end = clock();
  double t = double(end-start) / double(CLOCKS_PER_SEC) / double(numMats);
  double rate = 1.0 / t;
  fprintf (stderr, "Rate is %1.3f matrix inversions per second.\n",
           rate);

  // Copy A^(-1) back to host memory Ainv; one matrix at a time
  // Calculate error of A^(-1)A from unit matrix I
  for (int mat=0; mat<numMats; mat++)
  {
    T Ainv[N*row_stride];
    callAndCheckError( cudaMemcpy(Ainv, Clist[mat], N * row_stride * sizeof(T),
                                  cudaMemcpyDeviceToHost), __LINE__ );

    double error = 0.0;
    for (int i=0; i<N; i++)
      for (int j=0; j<N; j++)
      {
        T val = 0.0;
        for (int k=0; k<N; k++)
          val += Ainv[i*row_stride+k] * A[mat*N*row_stride+k*row_stride+j];
        double diff = (i==j) ? (1.0f - std::real(val)) : std::real(val);
        error += diff * diff;
      }
    fprintf (stderr, "error = %1.8e\n", sqrt(error/(double)(N*N)));
  }

  // Finalize cublas
  callAndCheckError( cublasDestroy(handle), __LINE__ );

  // Free resources on both host and device
  for (int mat=0; mat<numMats; mat++)
  {
    cudaFree(Alist[mat]);
    cudaFree(Clist[mat]);
    cudaFree(AWorklist[mat]);
    cudaFree(CWorklist[mat]);
  }
  cudaFree(Alist_d);
  cudaFree(Clist_d);
  cudaFree(AWorklist_d);
  cudaFree(CWorklist_d);
  free(Alist);
  free(Clist);
  free(AWorklist);
  free(CWorklist);
  free(A);

  // Reset device. Required for memory leak debugging
  cudaDeviceReset();

}


int main(int argc, char** argv)
{
  int matSize = 0;
  int numMats = 0;

  if (argc == 3) {
    matSize = atoi(argv[1]);
    numMats = atoi(argv[2]);
  }
  else {
    printf("Usage: ./cuda_inverse [matrix size] [number of matrices]\n");
    exit(1);
  }

#ifndef QMC_COMPLEX
  test_cublas_inverse<double>(matSize, numMats);
  test_cublas_inverse<float>(matSize, numMats);
#else
  test_cublas_inverse<std::complex<double> >(matSize, numMats);
  test_cublas_inverse<std::complex<float> >(matSize, numMats);
#endif

  return 0;
}

#endif
