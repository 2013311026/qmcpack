//////////////////////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source License.
// See LICENSE file in top directory for details.
//
// Copyright (c) 2016 Jeongnim Kim and QMCPACK developers.
//
// File developed by: Ken Esler, kpesler@gmail.com, StoneRidge Inc.
//		      Jeremy McMinnis, jmcminis@gmail.com, Navar Inc. 
//    		      Jeongnim Kim, jeongnim.kim@intel.com, Intel Inc.
//                    Ying Wai Li, yingwaili@ornl.gov, Oak Ridge National Laboratory
//                    Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//
// File created by:  Ken Esler, kpesler@gmail.com, StoneRidge Inc.
//////////////////////////////////////////////////////////////////////////////////////




// ============ Matrix inversion using cuBLAS library ============ //
//
// To compile as a standalone test program:
//
// 1. Make sure libcublas.so is in the search path
// 2. cd to build/ directory
// 3. nvcc -o cuda_inverse -arch=sm_35 -lcublas -DCUDA_TEST_MAIN
//         ../src/Numerics/CUDA/cuda_inverse.cu
//
// =============================================================== //

#include <cstdio>
#include <unistd.h>
#include <sstream>
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CONVERT_BS 256
#define INVERSE_BS 16


void
callAndCheckError (hipError_t cudaFunc, const int line)
{
  if (cudaFunc != hipSuccess)
  {
    fprintf(stderr, "CUDA error in %s, line %d \n", __FILE__, line); 
    fprintf(stderr, "CUDA error message : %s \n", hipGetErrorString(cudaFunc));
    fflush(stderr);
    abort();
  }
}

void
callAndCheckError(hipblasStatus_t cublasFunc, const int line)
{
  if (cublasFunc != HIPBLAS_STATUS_SUCCESS)
  {
    fprintf(stderr, "CUBLAS error in %s, line %d \n", __FILE__, line);
    fprintf(stderr, "CUBLAS error message: ");
    switch (cublasFunc)
    {
       case HIPBLAS_STATUS_NOT_INITIALIZED:
         fprintf(stderr, "CUBLAS_STATUS_NOT_INITIALIZED\n");
         break;
       case HIPBLAS_STATUS_ALLOC_FAILED:
         fprintf(stderr, "CUBLAS_STATUS_ALLOC_FAILED\n");
         break;
       case HIPBLAS_STATUS_INVALID_VALUE:
         fprintf(stderr, "CUBLAS_STATUS_INVALID_VALUE\n");
         break;
       case HIPBLAS_STATUS_ARCH_MISMATCH:
         fprintf(stderr, "CUBLAS_STATUS_ARCH_MISMATCH\n");
         break;
       case HIPBLAS_STATUS_MAPPING_ERROR:
         fprintf(stderr, "CUBLAS_STATUS_MAPPING_ERROR\n");
         break;
       case HIPBLAS_STATUS_EXECUTION_FAILED:
         fprintf(stderr, "CUBLAS_STATUS_EXECUTION_FAILED\n");
         break;
       case HIPBLAS_STATUS_INTERNAL_ERROR:
         fprintf(stderr, "CUBLAS_STATUS_INTERNAL_ERROR\n");
         break;
#if (CUDA_VERSION >= 6050)
       case HIPBLAS_STATUS_NOT_SUPPORTED:
         fprintf(stderr, "CUBLAS_STATUS_NOT_SUPPORTED\n");
         break;
       case HIPBLAS_STATUS_UNKNOWN:
         fprintf(stderr, "CUBLAS_STATUS_LICENSE_ERROR\n");
         break;
#endif
       default:
         fprintf(stderr, "unknown\n");
    }
    fflush(stderr);
    abort();
  }
}

// Convert matrix elements from one type (Tsrc) in the source matrix to
// another type (Tdest) and put them in the destination matrix
// (assumed src and dest have the same dimensions)
template <typename Tdest, typename Tsrc>
__global__ void
convert (Tdest **dest_list, Tsrc **src_list, int len)
{
  __shared__ Tsrc *mysrc;
  __shared__ Tdest *mydest;
  if (threadIdx.x == 0)
  {
    mysrc = src_list[blockIdx.y];
    mydest = dest_list[blockIdx.y];
  }
  __syncthreads();
  int i = blockIdx.x * CONVERT_BS + threadIdx.x;
  if (i < len)
    mydest[i] = (Tdest) mysrc[i];
}

// Two matrix inversion functions
// 1. for float matrices
//    useHigherPrecision = false --> single precision operations
//    useHigherPrecision = true  --> double precision operations
void
cublas_inverse (hipblasHandle_t handle, 
                float *Alist_d[], float *Ainvlist_d[],
                float *AWorklist_d[], float *AinvWorklist_d[],
                int N, int rowStride, int numMats,
                bool useHigherPrecision)
{

  // Info array tells if a matrix inversion is successful
  // = 0 : successful
  // = k : U(k,k) = 0; inversion failed 
  int *infoArray;
  callAndCheckError( hipMalloc((void**) &infoArray, numMats * sizeof(int)), __LINE__ ); 

  // If double precision operations are desired...
  if (useHigherPrecision)
  {

    // (i)   convert elements in Alist from float to double, put them in AWorklist
    dim3 dimBlockConvert (CONVERT_BS);
    dim3 dimGridConvert ((N*rowStride + (CONVERT_BS-1)) / CONVERT_BS, numMats);
    convert <<< dimGridConvert, dimBlockConvert >>> ((double**)AWorklist_d, Alist_d, N*rowStride);

    // (ii)  call cublas to do matrix inversion
    //       LU decomposition
    callAndCheckError( hipblasDgetrfBatched( handle, N, (double**)AWorklist_d, rowStride, NULL, 
                                            infoArray, numMats), __LINE__ );
  
    //       Inversion
#if (CUDA_VERSION >= 6050)
    callAndCheckError( hipblasDgetriBatched( handle, N, (const double**)AWorklist_d, rowStride, NULL, 
                                            (double**)AinvWorklist_d, rowStride, infoArray, numMats), __LINE__ );
#else
    callAndCheckError( cublasDgetriBatched( handle, N, (double**)AWorklist_d, rowStride, NULL, 
                                            (double**)AinvWorklist_d, rowStride, infoArray, numMats), __LINE__ );
#endif

    // (iii) convert results back to single precision
    convert <<< dimGridConvert, dimBlockConvert >>> (Ainvlist_d, (double**)AinvWorklist_d, N*rowStride);

  }
  // else, carry out single precision operations
  else
  {
    // Call cublas to do matrix inversion
    // LU decomposition
    callAndCheckError( hipblasSgetrfBatched( handle, N, Alist_d, rowStride, NULL, 
                                            infoArray, numMats), __LINE__ );
  
    // Inversion
#if (CUDA_VERSION >= 6050)
    callAndCheckError( hipblasSgetriBatched( handle, N, (const float**) Alist_d, rowStride, NULL, 
                                            Ainvlist_d, rowStride, infoArray, numMats), __LINE__ );
#else
    callAndCheckError( cublasSgetriBatched( handle, N, Alist_d, rowStride, NULL, 
                                            Ainvlist_d, rowStride, infoArray, numMats), __LINE__ );
#endif
  }

  hipDeviceSynchronize();

  // Free resources
  hipFree(infoArray);

}

// 2. for double matrices
void
cublas_inverse (hipblasHandle_t handle, 
                double *Alist_d[], double *Ainvlist_d[],
                double *AWorklist_d[], double *AinvWorklist_d[],
                int N, int rowStride, int numMats,
                bool useHigherPrecision)
{

  // Info array tells if a matrix inversion is successful
  // = 0 : successful
  // = k : U(k,k) = 0; inversion failed 
  int *infoArray;
  callAndCheckError( hipMalloc((void**) &infoArray, numMats * sizeof(int)), __LINE__ );

  // (i)   copy all the elements of Alist to AWorklist
  dim3 dimBlockConvert (CONVERT_BS);
  dim3 dimGridConvert ((N*rowStride + (CONVERT_BS-1)) / CONVERT_BS, numMats);
  convert <<< dimGridConvert, dimBlockConvert >>> (AWorklist_d, Alist_d, N*rowStride);
  
  // (ii)  call cublas functions to do inversion
  //       LU decomposition
  callAndCheckError( hipblasDgetrfBatched( handle, N, AWorklist_d, rowStride, NULL, 
                                          infoArray, numMats), __LINE__ );

  //       Inversion
#if (CUDA_VERSION >= 6050)
  callAndCheckError( hipblasDgetriBatched( handle, N, (const double**) AWorklist_d, rowStride, NULL, 
                                          Ainvlist_d, rowStride, infoArray, numMats), __LINE__ );
#else
  callAndCheckError( cublasDgetriBatched( handle, N, AWorklist_d, rowStride, NULL, 
                                          Ainvlist_d, rowStride, infoArray, numMats), __LINE__ );
#endif

  hipDeviceSynchronize();

  hipFree(infoArray);

}


//////////////////////////////////////////////////////
//                  Test routines                   //
//////////////////////////////////////////////////////

#ifdef CUDA_TEST_MAIN

template<typename T>
void
test_cublas_inverse(int matSize, int numMats)
{

  // Initialize cublas
  cublasHandle_t handle;
  callAndCheckError( cublasCreate(&handle), __LINE__);

  srand48((long) 12394);
  int N = matSize;
  int row_stride = (matSize+15) / 16 * 16;
  T **Alist, **AWorklist;
  T **Alist_d, **AWorklist_d;
  T **Clist, **CWorklist;
  T **Clist_d, **CWorklist_d;

  // Allocate arrays of pointers (one set on host, one set on device)
  // pointing to the starting address (on device) of each matrix and its buffer
  // (similar to DiracDeterminantCUDA)
  Alist = (T**) malloc(numMats * sizeof(T*));
  callAndCheckError( cudaMalloc((void**) &Alist_d, numMats * sizeof(T*)), __LINE__ );

  AWorklist = (T**) malloc(numMats * sizeof(T*));
  callAndCheckError( cudaMalloc((void**) &AWorklist_d, numMats * sizeof(T*)), __LINE__ );

  Clist = (T**) malloc(numMats * sizeof(T*));
  callAndCheckError( cudaMalloc((void**) &Clist_d, numMats * sizeof(T*)), __LINE__ );

  CWorklist = (T**) malloc(numMats * sizeof(T*));
  callAndCheckError( cudaMalloc((void**) &CWorklist_d, numMats * sizeof(T*)), __LINE__ );

  // Generate matrices filled with random numbers
  T* A = (T*) malloc(sizeof(T) * numMats * N * row_stride);

  for (int j=0; j<numMats; j++)
    for (int i=0; i<N*row_stride; i++)
      A[j*N*row_stride+i] = 1.0 * (drand48() - 0.5);

  // Allocate memory on device for each matrix
  for (int mat=0; mat<numMats; mat++)
  {
    callAndCheckError( cudaMalloc((void**) &(Alist[mat]), N * row_stride * sizeof(T)), __LINE__ );

    callAndCheckError( cudaMemcpyAsync(Alist[mat], &A[mat*N*row_stride],
                                       N * row_stride * sizeof(T),
                                       cudaMemcpyHostToDevice), __LINE__ );

    callAndCheckError( cudaMalloc((void**) &(AWorklist[mat]), 2 * N * row_stride * sizeof(T)), __LINE__ );

    callAndCheckError( cudaMalloc((void**) &(Clist[mat]), N * row_stride * sizeof(T)), __LINE__ );

    callAndCheckError( cudaMalloc((void**) &(CWorklist[mat]), 2 * N * row_stride * sizeof(T)), __LINE__ );
  }

  // Copy the starting address of each matrix
  callAndCheckError( cudaMemcpyAsync (Alist_d, Alist, numMats * sizeof(T*),
                                      cudaMemcpyHostToDevice), __LINE__ ); 

  callAndCheckError( cudaMemcpyAsync (AWorklist_d, AWorklist, numMats * sizeof(T*),
                                      cudaMemcpyHostToDevice), __LINE__ );

  callAndCheckError( cudaMemcpyAsync (Clist_d, Clist, numMats * sizeof(T*),
                                      cudaMemcpyHostToDevice), __LINE__ );

  callAndCheckError( cudaMemcpyAsync (CWorklist_d, CWorklist, numMats * sizeof(T*),
                                      cudaMemcpyHostToDevice), __LINE__ );

  cudaDeviceSynchronize();
 
  clock_t start = clock();
 
  // Call cublas functions to do inversion
  cublas_inverse (handle, Alist_d, Clist_d, AWorklist_d, CWorklist_d, N, row_stride, numMats, true);
  cudaDeviceSynchronize();
  
  clock_t end = clock();
  double t = double(end-start) / double(CLOCKS_PER_SEC) / double(numMats);
  double rate = 1.0 / t;
  fprintf (stderr, "Rate is %1.3f matrix inversions per second.\n",
           rate);

  // Copy A^(-1) back to host memory Ainv; one matrix at a time
  // Calculate error of A^(-1)A from unit matrix I
  for (int mat=0; mat<numMats; mat++)
  {
    T Ainv[N*row_stride];
    callAndCheckError( cudaMemcpy(Ainv, Clist[mat], N * row_stride * sizeof(T),
                                  cudaMemcpyDeviceToHost), __LINE__ );

    double error = 0.0;
    for (int i=0; i<N; i++)
      for (int j=0; j<N; j++)
      {
        double val = 0.0;
        for (int k=0; k<N; k++)
          val += Ainv[i*row_stride+k] * A[mat*N*row_stride+k*row_stride+j];
        double diff = (i==j) ? (1.0f - val) : val;
        error += diff * diff;
      }
    fprintf (stderr, "error = %1.8e\n", sqrt(error/(double)(N*N)));
  }

  // Finalize cublas
  callAndCheckError( cublasDestroy(handle), __LINE__ );

  // Free resources on both host and device
  for (int mat=0; mat<numMats; mat++)
  {
    cudaFree(Alist[mat]);
    cudaFree(Clist[mat]);
    cudaFree(AWorklist[mat]);
    cudaFree(CWorklist[mat]);
  }
  cudaFree(Alist_d);
  cudaFree(Clist_d);
  cudaFree(AWorklist_d);
  cudaFree(CWorklist_d);
  free(Alist);
  free(Clist);
  free(AWorklist);
  free(CWorklist);
  free(A);

  // Reset device. Required for memory leak debugging
  cudaDeviceReset();

}


int main(int argc, char** argv)
{
  int matSize = 0;
  int numMats = 0;

  if (argc == 3) {
    matSize = atoi(argv[1]);
    numMats = atoi(argv[2]);
  }
  else {
    printf("Usage: ./cuda_inverse [matrix size] [number of matrices]\n");
    exit(1);
  }

  test_cublas_inverse<double>(matSize, numMats);
  test_cublas_inverse<float>(matSize, numMats);

  return 0;
}

#endif
