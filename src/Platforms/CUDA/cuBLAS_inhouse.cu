#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source License.
// See LICENSE file in top directory for details.
//
// Copyright (c) 2020 QMCPACK developers.
//
// File developed by: Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//
// File created by: Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//////////////////////////////////////////////////////////////////////////////////////


#include "Platforms/CUDA/cuBLAS_inhouse.hpp"
#include <stdexcept>
#include <hip/hip_complex.h>

namespace qmcplusplus
{
namespace cuBLAS_inhouse
{
template<typename T, int ROWBS, int COLBS>
__global__ void gemvT_batched_kernel(const int m, // number of columns in row major A
                                     const int n, // number of rows in row major A
                                     const T* __restrict__ alpha,
                                     const T* const A[],
                                     const int lda,
                                     const T* const x[],
                                     const int incx,
                                     const T* __restrict__ beta,
                                     T* const y[],
                                     const int incy)
{
  static_assert(ROWBS <= COLBS, "Row block size must not be larger than column block size!");

  __shared__ T sum[ROWBS][COLBS];
  __shared__ T x_part[COLBS];

  const int tid = threadIdx.x;
  for (int i = 0; i < ROWBS; i++)
    sum[i][tid] = T(0.0);

  const T* __restrict__ A_iw = A[blockIdx.x];
  const T* __restrict__ x_iw = x[blockIdx.x];

  const int row_begin = blockIdx.y * ROWBS;
  const int row_max   = (n - row_begin) < ROWBS ? (n - row_begin) : ROWBS;

  const int num_col_blocks = (m + COLBS - 1) / COLBS;
  for (int ib = 0; ib < num_col_blocks; ib++)
  {
    const int col_id = ib * COLBS + tid;
    if (col_id < m)
      x_part[tid] = x_iw[col_id * incx];
    for (int row_id = row_begin; row_id < row_begin + row_max; row_id++)
      if (col_id < m)
        sum[row_id - row_begin][tid] += x_part[tid] * A_iw[row_id * lda + col_id];
  }

  for (int iend = COLBS / 2; iend > 0; iend /= 2)
  {
    __syncthreads();
    for (int irow = 0; irow < row_max; irow++)
      if (tid < iend)
        sum[irow][tid] += sum[irow][tid + iend];
  }

  __syncthreads();
  T* __restrict__ y_iw = y[blockIdx.x];
  if (tid < row_max)
    y_iw[(row_begin + tid) * incy] = alpha[blockIdx.x] * sum[tid][0] + beta[blockIdx.x] * y_iw[(row_begin + tid) * incy];
}

template<typename T, int ROWBS>
__global__ void gemvN_batched_kernel(const int m, // number of columns in row major A
                                     const int n, // number of rows in row major A
                                     const T* __restrict__ alpha,
                                     const T* const A[],
                                     const int lda,
                                     const T* const x[],
                                     const int incx,
                                     const T* __restrict__ beta,
                                     T* const y[],
                                     const int incy)
{
  const T* __restrict__ A_iw = A[blockIdx.x];
  const T* __restrict__ x_iw = x[blockIdx.x];
  T* __restrict__ y_iw = y[blockIdx.x];

  const int tid = threadIdx.x;
  const int row_begin = blockIdx.y * ROWBS;

  T sum(0);
  for (int col_id = 0; col_id < n; col_id++)
    sum += A_iw[col_id * lda + row_begin + tid] * x_iw[col_id * incx];
  y_iw[(row_begin + tid) * incy] = alpha[blockIdx.x] * sum + beta[blockIdx.x] * y_iw[(row_begin + tid) * incy];
}

template<typename T>
cuBLAS_inhouse_status gemv_batched_impl(cuBLAS_inhouse_handle& handle,
                                        const char trans,
                                        const int m,
                                        const int n,
                                        const T* alpha,
                                        const T* const A[],
                                        const int lda,
                                        const T* const x[],
                                        const int incx,
                                        const T* beta,
                                        T* const y[],
                                        const int incy,
                                        const int batch_count)
{
  if (batch_count == 0 || m == 0 || n == 0)
    return hipSuccess;

  if (trans == 'T')
  {
    const int ROWBS          = 4;
    const int COLBS          = 64;
    const int num_row_blocks = (n + ROWBS - 1) / ROWBS;
    dim3 dimBlock(COLBS);
    dim3 dimGrid(batch_count, num_row_blocks);
    gemvT_batched_kernel<T, ROWBS, COLBS><<<dimGrid, dimBlock, 0, handle>>>(m, n, alpha, A, lda, x, incx, beta, y, incy);
  }
  else
  {
    const int ROWBS          = 64;
    const int num_row_blocks = (m + ROWBS - 1) / ROWBS;
    dim3 dimBlock(ROWBS);
    dim3 dimGrid(batch_count, num_row_blocks);
    gemvN_batched_kernel<T, ROWBS><<<dimGrid, dimBlock, 0, handle>>>(m, n, alpha, A, lda, x, incx, beta, y, incy);
  }
  return hipPeekAtLastError();
}

cuBLAS_inhouse_status gemv_batched(cuBLAS_inhouse_handle& handle,
                                   const char trans,
                                   const int m,
                                   const int n,
                                   const float* alpha,
                                   const float* const A[],
                                   const int lda,
                                   const float* const x[],
                                   const int incx,
                                   const float* beta,
                                   float* const y[],
                                   const int incy,
                                   const int batch_count)
{
  return gemv_batched_impl(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy, batch_count);
}

cuBLAS_inhouse_status gemv_batched(cuBLAS_inhouse_handle& handle,
                                   const char trans,
                                   const int m,
                                   const int n,
                                   const double* alpha,
                                   const double* const A[],
                                   const int lda,
                                   const double* const x[],
                                   const int incx,
                                   const double* beta,
                                   double* const y[],
                                   const int incy,
                                   const int batch_count)
{
  return gemv_batched_impl(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy, batch_count);
}

cuBLAS_inhouse_status gemv_batched(cuBLAS_inhouse_handle& handle,
                                   const char trans,
                                   const int m,
                                   const int n,
                                   const std::complex<float>* alpha,
                                   const std::complex<float>* const A[],
                                   const int lda,
                                   const std::complex<float>* const x[],
                                   const int incx,
                                   const std::complex<float>* beta,
                                   std::complex<float>* const y[],
                                   const int incy,
                                   const int batch_count)
{
  //return gemv_batched_impl(handle, trans, m, n, (const hipComplex*)alpha, (const hipComplex* const*)A, lda, (const hipComplex* const*)x, incx, (const hipComplex*)beta, (hipComplex* const*)y, incy, batch_count);
  return hipSuccess;
}

cuBLAS_inhouse_status gemv_batched(cuBLAS_inhouse_handle& handle,
                                   const char trans,
                                   const int m,
                                   const int n,
                                   const std::complex<double>* alpha,
                                   const std::complex<double>* const A[],
                                   const int lda,
                                   const std::complex<double>* const x[],
                                   const int incx,
                                   const std::complex<double>* beta,
                                   std::complex<double>* const y[],
                                   const int incy,
                                   const int batch_count)
{
  //return gemv_batched_impl(handle, trans, m, n, (const hipDoubleComplex*)alpha, (const hipDoubleComplex* const*)A, lda, (const hipDoubleComplex* const*)x, incx, (const hipDoubleComplex*)beta, (hipDoubleComplex* const*)y, incy, batch_count);
  return hipSuccess;
}


template<typename T, int ROWBS, int COLBS>
__global__ void ger_batched_kernel(const int m, // number of columns in row major A
                                   const int n, // number of rows in row major A
                                   const T* __restrict__ alpha,
                                   const T* const x[],
                                   const int incx,
                                   const T* const y[],
                                   const int incy,
                                   T* const A[],
                                   const int lda)
{
  const int iw = blockIdx.x;
  const T* __restrict__ x_iw = x[iw];
  const T* __restrict__ y_iw = y[iw];
  T* __restrict__ A_iw = A[iw];

  const int row_begin = blockIdx.y * ROWBS;
  const int row_end   = (row_begin + ROWBS) < n ? (row_begin + ROWBS) : n;
  const int tid = threadIdx.x;
  const int col_id = blockIdx.z * COLBS + tid;

  __shared__ T x_part[COLBS];
  if (col_id < m)
    x_part[tid] = x_iw[col_id * incx];

  for (int row_id = row_begin; row_id < row_end; row_id++)
    if (col_id < m)
      A_iw[row_id * lda + col_id] += alpha[iw] * x_part[tid] * y_iw[row_id * incy];
}

template<typename T>
cuBLAS_inhouse_status ger_batched_impl(cuBLAS_inhouse_handle& handle,
                                       const int m,
                                       const int n,
                                       const T* alpha,
                                       const T* const x[],
                                       const int incx,
                                       const T* const y[],
                                       const int incy,
                                       T* const A[],
                                       const int lda,
                                       const int batch_count)
{
  if (batch_count == 0 || m == 0 || n == 0)
    return hipSuccess;

  const int ROWBS          = 16;
  const int COLBS          = 64;
  const int num_row_blocks = (n + ROWBS - 1) / ROWBS;
  const int num_col_blocks = (m + COLBS - 1) / COLBS;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count, num_row_blocks, num_col_blocks);
  ger_batched_kernel<T, ROWBS, COLBS><<<dimGrid, dimBlock, 0, handle>>>(m, n, alpha, x, incx, y, incy, A, lda);
  return hipPeekAtLastError();
}

cuBLAS_inhouse_status ger_batched(cuBLAS_inhouse_handle& handle,
                                  const int m,
                                  const int n,
                                  const float* alpha,
                                  const float* const x[],
                                  const int incx,
                                  const float* const y[],
                                  const int incy,
                                  float* const A[],
                                  const int lda,
                                  const int batch_count)
{
  return ger_batched_impl(handle, m, n, alpha, x, incx, y, incy, A, lda, batch_count);
}

cuBLAS_inhouse_status ger_batched(cuBLAS_inhouse_handle& handle,
                                  const int m,
                                  const int n,
                                  const double* alpha,
                                  const double* const x[],
                                  const int incx,
                                  const double* const y[],
                                  const int incy,
                                  double* const A[],
                                  const int lda,
                                  const int batch_count)
{
  return ger_batched_impl(handle, m, n, alpha, x, incx, y, incy, A, lda, batch_count);
}

cuBLAS_inhouse_status ger_batched(cuBLAS_inhouse_handle& handle,
                                  const int m,
                                  const int n,
                                  const std::complex<float>* alpha,
                                  const std::complex<float>* const x[],
                                  const int incx,
                                  const std::complex<float>* const y[],
                                  const int incy,
                                  std::complex<float>* const A[],
                                  const int lda,
                                  const int batch_count)
{
  //return ger_batched_impl(handle, m, n, alpha, x, incx, y, incy, A, lda, batch_count);
  return hipSuccess;
}

cuBLAS_inhouse_status ger_batched(cuBLAS_inhouse_handle& handle,
                                  const int m,
                                  const int n,
                                  const std::complex<double>* alpha,
                                  const std::complex<double>* const x[],
                                  const int incx,
                                  const std::complex<double>* const y[],
                                  const int incy,
                                  std::complex<double>* const A[],
                                  const int lda,
                                  const int batch_count)
{
  //return ger_batched_impl(handle, m, n, alpha, x, incx, y, incy, A, lda, batch_count);
  return hipSuccess;
}

template<typename T, int COLBS>
__global__ void copy_batched_kernel(const int n,
                                    const T* const in[],
                                    T* const out[])
{
  const int iw                 = blockIdx.x;
  const T* __restrict__ in_iw  = in[iw];
  T* __restrict__ out_iw       = out[iw];

  const int col_id = blockIdx.y * COLBS + threadIdx.x;
  out_iw[col_id] = in_iw[col_id];
}

template<typename T>
cuBLAS_inhouse_status copy_batched_impl(hipStream_t& hstream,
                                   const int n,
                                   const T* const in[],
                                   const int incx,
                                   T* const out[],
                                   const int incy,
                                   const int batch_count)
{
  if (batch_count == 0 || n == 0)
    return hipSuccess;

  if (incx != 1 || incy != 1)
    throw std::runtime_error("incx !=1 or incy != 1 are not implemented in cuBLAS_inhouse::copy_batched_impl!");

  const int COLBS = 128;
  const int num_col_blocks = (n + COLBS - 1) / COLBS;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count, num_col_blocks);
  copy_batched_kernel<T, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(n, in, out);
  return hipPeekAtLastError();
}

cuBLAS_inhouse_status copy_batched(hipStream_t& hstream,
                                   const int n,
                                   const float* const in[],
                                   const int incx,
                                   float* const out[],
                                   const int incy,
                                   const int batch_count)
{
  return copy_batched_impl(hstream, n, in, incx, out, incy, batch_count);
}

cuBLAS_inhouse_status copy_batched(hipStream_t& hstream,
                                   const int n,
                                   const double* const in[],
                                   const int incx,
                                   double* const out[],
                                   const int incy,
                                   const int batch_count)
{
  return copy_batched_impl(hstream, n, in, incx, out, incy, batch_count);
}

cuBLAS_inhouse_status copy_batched(hipStream_t& hstream,
                                   const int n,
                                   const std::complex<float>* const in[],
                                   const int incx,
                                   std::complex<float>* const out[],
                                   const int incy,
                                   const int batch_count)
{
  //return copy_batched_impl(hstream, n, in, incx, out, incy, batch_count);
  return hipSuccess;
}

cuBLAS_inhouse_status copy_batched(hipStream_t& hstream,
                                   const int n,
                                   const std::complex<double>* const in[],
                                   const int incx,
                                   std::complex<double>* const out[],
                                   const int incy,
                                   const int batch_count)
{
  //return copy_batched_impl(hstream, n, in, incx, out, incy, batch_count);
  return hipSuccess;
}

// Xscal_batched
template<typename T, int COLBS>
__global__ void scal_batched_kernel(const int n,
                                    const T* const scale,
                                    T* const inout[],
                                    const int inc)
{
  const int iw             = blockIdx.x;
  T* __restrict__ inout_iw = inout[iw];
  const T scale_iw = scale[iw];

  const int col_id = blockIdx.y * COLBS + threadIdx.x;
  inout_iw[col_id * inc] *= scale_iw;
}

template<typename T>
cuBLAS_inhouse_status scal_batched_impl(hipStream_t& hstream,
                                   const int n,
                                   const T* const scale,
                                   T* const inout[],
                                   const int inc,
                                   const int batch_count)
{
  if (batch_count == 0 || n == 0)
    return hipSuccess;

  const int COLBS = 128;
  const int num_col_blocks = (n + COLBS - 1) / COLBS;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count, num_col_blocks);
  scal_batched_kernel<T, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(n, scale, inout, inc);
  return hipPeekAtLastError();
}

cuBLAS_inhouse_status scal_batched(hipStream_t& hstream,
                                   const int n,
                                   const float* const scale,
                                   float* const inout[],
                                   const int inc,
                                   const int batch_count)
{
  return scal_batched_impl(hstream, n, scale, inout, inc, batch_count);
}

cuBLAS_inhouse_status scal_batched(hipStream_t& hstream,
                                   const int n,
                                   const double* const scale,
                                   double* const inout[],
                                   const int inc,
                                   const int batch_count)
{
  return scal_batched_impl(hstream, n, scale, inout, inc, batch_count);
}

cuBLAS_inhouse_status scal_batched(hipStream_t& hstream,
                                   const int n,
                                   const std::complex<float>* const scale,
                                   std::complex<float>* const inout[],
                                   const int inc,
                                   const int batch_count)
{
  //return scal_batched_impl(hstream, n, scale, inout, inc, batch_count);
  return hipSuccess;
}

cuBLAS_inhouse_status scal_batched(hipStream_t& hstream,
                                   const int n,
                                   const std::complex<double>* const scale,
                                   std::complex<double>* const inout[],
                                   const int inc,
                                   const int batch_count)
{
  //return scal_batched_impl(hstream, n, scale, inout, inc, batch_count);
  return hipSuccess;
}

// Xdot_batched
template<typename T, int COLBS>
__global__ void dot_batched_kernel(const int n,
                                   const T* const x[],
                                   const int incx,
                                   const T* const y[],
                                   const int incy,
                                   T* const res)
{
  const int tid            = threadIdx.x;
  const int iw             = blockIdx.x;
  const T* __restrict__ x_iw = x[iw];
  const T* __restrict__ y_iw = y[iw];

  __shared__ T sum[COLBS];
  sum[tid] = T(0);
  const int num_col_blocks = (n + COLBS - 1) / COLBS;
  for (int ib = 0; ib < num_col_blocks; ib++)
  {
    const int col_id = ib * COLBS + tid;
    sum[col_id] += x_iw[col_id * incx] * y_iw[col_id * incy];
  }

  for (int iend = COLBS / 2; iend > 0; iend /= 2)
  {
    __syncthreads();
    if (tid < iend)
      sum[tid] += sum[tid + iend];
  }

  if (tid == 0)
    res[iw] = sum[0];
}

template<typename T>
cuBLAS_inhouse_status dot_batched_impl(hipStream_t& hstream,
                                       const int n,
                                       const T* const x[],
                                       const int incx,
                                       const T* const y[],
                                       const int incy,
                                       T* const res,
                                       const int batch_count)
{
  if (batch_count == 0 || n == 0)
    return hipSuccess;

  const int COLBS = 128;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count);
  dot_batched_kernel<T, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(n, x, incx, y, incy, res);
  return hipPeekAtLastError();
}

cuBLAS_inhouse_status dot_batched(hipStream_t& hstream,
                                  const int n,
                                  const float* const x[],
                                  const int incx,
                                  const float* const y[],
                                  const int incy,
                                  float* const res,
                                  const int batch_count)
{
  return dot_batched_impl(hstream, n, x, incx, y, incy, res, batch_count);
}

cuBLAS_inhouse_status dot_batched(hipStream_t& hstream,
                                  const int n,
                                  const double* const x[],
                                  const int incx,
                                  const double* const y[],
                                  const int incy,
                                  double* const res,
                                  const int batch_count)
{
  return dot_batched_impl(hstream, n, x, incx, y, incy, res, batch_count);
}

cuBLAS_inhouse_status dot_batched(hipStream_t& hstream,
                                  const int n,
                                  const std::complex<float>* const x[],
                                  const int incx,
                                  const std::complex<float>* const y[],
                                  const int incy,
                                  std::complex<float>* const res,
                                  const int batch_count)
{
  //return dot_batched_impl(hstream, n, x, incx, y, incy, res, batch_count);
  return hipSuccess;
}

cuBLAS_inhouse_status dot_batched(hipStream_t& hstream,
                                  const int n,
                                  const std::complex<double>* const x[],
                                  const int incx,
                                  const std::complex<double>* const y[],
                                  const int incy,
                                  std::complex<double>* const res,
                                  const int batch_count)
{
  //return dot_batched_impl(hstream, n, x, incx, y, incy, res, batch_count);
  return hipSuccess;
}

} // namespace cuBLAS_inhouse
} // namespace qmcplusplus
