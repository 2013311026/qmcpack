//////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source
// License.  See LICENSE file in top directory for details.
//
// Copyright (c) 2016 Jeongnim Kim and QMCPACK developers.
//
// File developed by:
//    Lawrence Livermore National Laboratory 
//
// File created by:
// Miguel A. Morales, moralessilva2@llnl.gov 
//    Lawrence Livermore National Laboratory 
////////////////////////////////////////////////////////////////////////////////

#include<cassert>
#include <complex>
#include<hip/hip_runtime.h>
#include "hiprand.h"
#include<hip/hip_runtime.h>
#include "AFQMC/Kernels/cuda_settings.h"
#include "AFQMC/Kernels/zero_complex_part.hpp"
#define QMC_CUDA 1

namespace kernels
{

void sampleGaussianRNG( double* V, int n, hiprandGenerator_t & gen) 
{
  qmc_cuda::cuda_check(hiprandGenerateNormalDouble(gen,V,n,0.0,1.0),
                                          "hiprandGenerateNormalDouble");
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void sampleGaussianRNG( float* V, int n, hiprandGenerator_t & gen) 
{
  qmc_cuda::cuda_check(curandGenerateNormalFloat(gen,V,n,0.0,1.0),
                                          "curandGenerateNormalFloat");
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void sampleGaussianRNG( std::complex<double>* V, int n, hiprandGenerator_t & gen) 
{
  qmc_cuda::cuda_check(hiprandGenerateNormalDouble(gen,
                        reinterpret_cast<double*>(V),2*n,0.0,1.0),
                                          "hiprandGenerateNormalDouble");
  qmc_cuda::cuda_check(hipDeviceSynchronize());
  // hack hack hack!!!
  kernels::zero_complex_part(n,V);
}

void sampleGaussianRNG( std::complex<float>* V, int n, hiprandGenerator_t & gen) 
{
  qmc_cuda::cuda_check(curandGenerateNormalFloat(gen,
                        reinterpret_cast<float*>(V),2*n,0.0,1.0),
                                          "curandGenerateNormalFloat");
  qmc_cuda::cuda_check(hipDeviceSynchronize());
  // hack hack hack!!!
  kernels::zero_complex_part(n,V);
} 

}
